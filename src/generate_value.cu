#include "hip/hip_runtime.h"
#include "cuda_put.hpp"

template <typename Data_t>
__global__ void assign(Data_t *ptr, int size, int var_idx);


template <>
__global__ void assign<double>(double *ptr, int size, int var_idx)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx<size) {
        ptr[idx] = idx + 0.01*var_idx;
    }
}

template <>
__global__ void assign<float>(float *ptr, int size, int var_idx)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx<size) {
        ptr[idx] = idx + 0.01*var_idx;
    }
}

hipError_t cuda_assign_double(MPI_Comm gcomm, double *ptr, int size, int var_idx)
{
    MPI_Comm_rank(gcomm, &rank);
    int dev_num, dev_rank;
    hipError_t cuda_status;
    hipDeviceProp_t dev_prop;
    cuda_status = hipGetDeviceCount(&dev_num);
    dev_rank = rank%dev_num;
    cuda_status = hipSetDevice(dev_rank);
    cuda_status = hipGetDeviceProperties(&dev_prop,dev_rank);

    int threadsPerBlock = dev_prop.maxThreadsPerBlock;
    int numBlocks = (grid_size + threadsPerBlock) / threadsPerBlock;

    assign<double><<<numBlocks, threadsPerBlock>>>(ptr, size, var_idx);

    return cuda_status
}

hipError_t cuda_assign_float(MPI_Comm gcomm, float *ptr, int size, int var_idx)
{
    MPI_Comm_rank(gcomm, &rank);
    int dev_num, dev_rank;
    hipError_t cuda_status;
    hipDeviceProp_t dev_prop;
    cuda_status = hipGetDeviceCount(&dev_num);
    dev_rank = rank%dev_num;
    cuda_status = hipSetDevice(dev_rank);
    cuda_status = hipGetDeviceProperties(&dev_prop,dev_rank);

    int threadsPerBlock = dev_prop.maxThreadsPerBlock;
    int numBlocks = (grid_size + threadsPerBlock) / threadsPerBlock;

    assign<float><<<numBlocks, threadsPerBlock>>>(ptr, size, var_idx);

    return cuda_status
}