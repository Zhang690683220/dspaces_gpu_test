
#include <hip/hip_runtime.h>

template <typename Data_t>
__global__ void assign(Data_t *ptr, int size, int var_idx);


template <>
__global__ void assign<double>(double *ptr, int size, int var_idx)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx<size) {
        ptr[idx] = idx + 0.01*var_idx;
    }
}

template <>
__global__ void assign<float>(float *ptr, int size, int var_idx)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx<size) {
        ptr[idx] = idx + 0.01*var_idx;
    }
}
