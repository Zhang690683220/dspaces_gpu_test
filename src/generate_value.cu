#include "hip/hip_runtime.h"
#include "cuda_put.hpp"

template <typename Data_t>
__global__ void assign(Data_t *ptr, int size, int var_idx);


template <>
__global__ void assign<double>(double *ptr, int size, int var_idx)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx<size) {
        ptr[idx] = idx + 0.01*var_idx;
    }
}

template <>
__global__ void assign<float>(float *ptr, int size, int var_idx)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx<size) {
        ptr[idx] = idx + 0.01*var_idx;
    }
}

hipError_t cuda_assign_double(int dev_rank, double *ptr, int size, int var_idx)
{
    hipError_t cuda_status;
    hipDeviceProp_t dev_prop;
    cuda_status = hipGetDeviceProperties(&dev_prop,dev_rank);

    int threadsPerBlock = dev_prop.maxThreadsPerBlock;
    int numBlocks = (size + threadsPerBlock) / threadsPerBlock;

    assign<double><<<numBlocks, threadsPerBlock>>>(ptr, size, var_idx);

    return cuda_status;
}

hipError_t cuda_assign_float(int dev_rank, float *ptr, int size, int var_idx)
{
    hipError_t cuda_status;
    hipDeviceProp_t dev_prop;
    cuda_status = hipGetDeviceProperties(&dev_prop,dev_rank);

    int threadsPerBlock = dev_prop.maxThreadsPerBlock;
    int numBlocks = (size + threadsPerBlock) / threadsPerBlock;

    assign<float><<<numBlocks, threadsPerBlock>>>(ptr, size, var_idx);

    return cuda_status;
}